#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cmath>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#define N 300000 // number of data points
#define D 30 // dimension of each data point
#define K 3 // number of nearest neighbors
#define BLOCK_SIZE_DIS 8 // number of threads per block for distance calculation
#define BLOCK_SIZE_KNN 1 // number of threads per block for knn calculation

class Timer {
public:
    Timer() {
        stopped=started=std::chrono::high_resolution_clock::now();
    };

    Timer& start() {
        started=std::chrono::high_resolution_clock::now();
        return *this;
    }

    Timer& stop() {
        stopped=std::chrono::high_resolution_clock::now();
        return *this;
    }

    double elapsed() {
        if(started!=stopped) {
            std::chrono::duration<double> elapsed = stopped - started;
            return elapsed.count();
        }
        return 0.0;
    }

    std::chrono::time_point<std::chrono::high_resolution_clock> started;
    std::chrono::time_point<std::chrono::high_resolution_clock> stopped;
};

//--------------------------------------------------------------------------------------//
//                                     CUDA Kernels                                     //
//--------------------------------------------------------------------------------------//

/**
 * This CUDA kernel calculates the Euclidean distance between a query point and a set of data points.
 *
 * input：dimension, num_points, data_input, query
 * @param dimension The dimension of each point (number of features).
 * @param num_points The number of data points to calculate distances for.
 * @param data_input An array containing the data points.
 * @param query An array containing the query point.
 *
 * output：distances
 * @param distances An array to store the calculated distances.
 *
 */
__global__ void euclidean_distance_kernel(float* distances, float* data_input, float* query, int dimension, int num_points) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        float sum = 0;
        for (int i = 0; i < dimension; i++) {
            float diff = data_input[idx * dimension + i] - query[i];
            sum += diff * diff;
        }
        distances[idx] = sqrt(sum);
    }
}

/**
 * This CUDA kernel use a merge sort to sort the distances and indices.
 * It is used in the main function to find the k nearest neighbors.
 *
 * input：num_points, ind_temp, dis_temp
 * @param num_points The number of distances to sort.
 * @param ind_temp An array temprarly storing the indices to pass to the next iteration.
 * @param dis_temp An array temprarly storing the distances to pass to the next iteration.
 *
 * output：indices, distances
 * @param indices An array to store the indices of the sorted distances.
 * @param distances An array to store the sorted distances.
 *
 * @NOTE: The number of threads per block must be 1. And the total number of the data points must be a power of 2.
 */
__global__ void mergeBlocks_kernel(int* indices, float* distances, int* ind_temp, float* dis_temp, int sortedsize) {
    int id = blockIdx.x;

    //index for distances
    int index1 = id * 2 * sortedsize;
    int endIndex1 = index1 + sortedsize;
    int index2 = endIndex1;
    int endIndex2 = index2 + sortedsize;
    int targetIndex = id * 2 * sortedsize;

    //pesudo index for indices
    //let the indices divide and merge like distances
    int index1_ind = index1;
    int endIndex1_ind = endIndex1;
    int index2_ind = index2;
    int endIndex2_ind = endIndex2;
    int targetIndex_ind = targetIndex;

    int done = 0;
    while (!done)
    {
        //if the first block is not finished and the second block is not finished
        if ((index1 == endIndex1) && (index2 < endIndex2)) {
            dis_temp[targetIndex++] = distances[index2++];
            ind_temp[targetIndex_ind++] = indices[index2_ind++];
        }

        //if the second block is finished and the first block is not finished
        else if ((index2 == endIndex2) && (index1 < endIndex1)) {
            dis_temp[targetIndex++] = distances[index1++];
            ind_temp[targetIndex_ind++] = indices[index1_ind++];
        }

        //if the first block is smaller than the second block
        else if (distances[index1] < distances[index2]) {
            dis_temp[targetIndex++] = distances[index1++];
            ind_temp[targetIndex_ind++] = indices[index1_ind++];
        }

        //else
        else {
            dis_temp[targetIndex++] = distances[index2++];
            ind_temp[targetIndex_ind++] = indices[index2_ind++];
        }

        //if both blocks are finished, merge done.
        if ((index1 == endIndex1) && (index2 == endIndex2))
            done = 1;
    }
}



//--------------------------------------------------------------------------------------//
//                                     CPU functions                                    //
//--------------------------------------------------------------------------------------//


/**
 * This CPU function calculates the Euclidean distance between a query point and a set of data points.
 *
 * input：dimension, num_points, data_input, query
 * @param dimension The dimension of each point (number of features).
 * @param num_points The number of data points to calculate distances for.
 * @param data_input An array containing the data points.
 * @param query An array containing the query point.
 *
 * output：distances
 * @param distances An array to store the calculated distances.
 *
 */
float euclidean_distance_cpu(std::vector<float> &vec1, std::vector<float> &vec2) {
    float distance = 0.0;
    for (int j = 0; j < vec1.size(); j++) {
        float diff = vec1[j] -vec2[j];
        distance += diff * diff;
    }

    return sqrt(distance);
}

/**
 * want some CPU function to find the k nearest neighbors
 * to compare running time with the GPU function in the main function
 *
 */

 /*DOESN'T SEEMS TO WORK*/

void knn_cpu(std::vector<std::pair<std::vector<float>, float>> *distances, std::vector<std::vector<float>> &trainSet, std::vector<float>& queryData) {
    for (auto  j : trainSet) {
        distances->push_back(std::make_pair( j, euclidean_distance_cpu(queryData,j)));
    }

    std::sort(distances->begin(), distances->end(),[](const std::pair<std::vector<float>, float>& p1, const std::pair<std::vector<float>, float>& p2){
        return p1.second < p2.second;
    });
}

//--------------------------------------------------------------------------------------//
//                                     main function                                    //
//--------------------------------------------------------------------------------------//


/**
 * main function to see if GPU and CPU functions work and compare running time
 */

int main() {

    /*============== Initialize and Memory Allocated ==============*/

    // (HOST) Define and allocate memory for data input
    float *data_input;
    data_input = (float *) malloc(N * D * sizeof(float));



    // (HOST) Initialize data input
    for (int i = 0; i < N * D; i++) {
        data_input[i] = (float) rand() / (float) RAND_MAX;
    }
    std::vector<std::vector<float> > data_input_vec(N, std::vector<float>(D));
    for(int i = 0; i < N*D; i++) {
        data_input_vec[i / D][i % D] = data_input[i];
    }
//    std::vector<float> data_input_vec(data_input, data_input + N * D);
    /*

    //using this to test if distance is calculated correctly and if the sorting works
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < D; j++) {
            data_input[i * D + j] = i+1;
        }
    }

    */

    // (HOST) Define and allocate memory for query data
    float *data_query;
    data_query = (float *) malloc(D * sizeof(float));

    // (HOST) Initialize query data
    for (int i = 0; i < D; i++) {
        data_query[i] = (float) rand() / (float) RAND_MAX;
    }
    std::vector<float> data_query_vec(data_query, data_query + D);

    /*
    //using this to test if distance is calculated correctly and if the sorting works
    for (int i = 0; i <D; i++) {
        data_query[i] = -1;
    }

    */

    // (DEVICE) Copy input data to device memory
    float *d_input;
    hipMalloc(&d_input, N * D * sizeof(float));
    hipMemcpy(d_input, data_input, N * D * sizeof(float), hipMemcpyHostToDevice);


    // (DEVICE) Copy query data to device memory
    float *d_query;
    hipMalloc(&d_query, D * sizeof(float));
    hipMemcpy(d_query, data_query, D * sizeof(float), hipMemcpyHostToDevice);

    // (DEVICE) Allocate memory for temporary distances and indices
    float *d_dis_temp;
    hipMalloc(&d_dis_temp, N * sizeof(float));
    int *d_ind_temp;
    hipMalloc(&d_ind_temp, N * sizeof(int));

    // (DEVICE) Allocate memory for indices
    int *d_indices;
    hipMalloc(&d_indices, N * sizeof(int));

    /*

    // Allocate memory for unsorted distances
    float* distances_unsorted = (float*)malloc(N * sizeof(float));
    hipMemcpy(distances_unsorted, d_distances, N * sizeof(float), hipMemcpyDeviceToHost);

    */

    // (DEVICE) Initialize indices and copy to device memory
    int *indices = (int *) malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) {
        indices[i] = i;
    }
    hipMemcpy(d_indices, indices, N * sizeof(int), hipMemcpyHostToDevice);

    // (DEVICE) Allocate memory for distances
    float *d_distances;
    hipMalloc(&d_distances, N * sizeof(float));

    // (HOST) Allocate memory for sorted distances and indices computed by GPU
    auto *distances_sorted = (float *) malloc(N * sizeof(float));
    int *indices_sorted = (int *) malloc(N * sizeof(int));

    // (HOST) Allocate memory for sorted distances and indices computed by CPU
    float * distances_sorted_CPU = (float *) malloc(N * sizeof(float));
    auto indices_sorted_CPU_vec = new std::vector<float>();


    /*============== GPU functions ==============*/

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start));

    // Compute distances between current query point and data input
    euclidean_distance_kernel <<<(N + BLOCK_SIZE_DIS - 1) / BLOCK_SIZE_DIS, BLOCK_SIZE_DIS >>>(d_distances, d_input,
                                                                                               d_query, D, N);


    // Sort distances and indices
    int blocks = (N + BLOCK_SIZE_KNN - 1) / BLOCK_SIZE_KNN / 2;
    int sortedsize = BLOCK_SIZE_KNN;
    while (blocks > 0) {
        mergeBlocks_kernel <<<blocks, 1 >>>(d_indices, d_distances, d_ind_temp, d_dis_temp, sortedsize);
        hipMemcpy(d_distances, d_dis_temp, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_indices, d_ind_temp, N * sizeof(int), hipMemcpyDeviceToDevice);
        blocks /= 2;
        sortedsize *= 2;
    }
    checkCudaErrors(hipEventRecord(stop));
    // Copy sorted distances and indices to host memory
    hipMemcpy(indices_sorted, d_indices, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(distances_sorted, d_distances, N * sizeof(float), hipMemcpyDeviceToHost);

//    std::vector<float> distances_sorted_vec_GPU(distances_sorted, distances_sorted + N);


    checkCudaErrors(hipEventSynchronize(stop));

    printf("============== GPU ==============\n\n");

    // Compute and print kernel execution time
    float kernel_time;
    checkCudaErrors(hipEventElapsedTime(&kernel_time, start, stop));
    printf("Kernel execution time\t\t\t: %f ms\n\n", kernel_time);

    // Print query point
    printf("Query point : ");
    for (int i = 0; i < D; i++) {
        printf("%f ", data_query[i]);
    }
    printf("\n\n");

    // Print sorted indices and distances
    printf("Find the indices of K nearest neighbors(GPU) : \n");
    for (int j = 0; j < K; j++) {
        printf("%d ", indices_sorted[j]);
    }
    printf("\n\n");

    printf("Find the distances of K nearest neighbors(GPU) : \n");
    for (int j = 0; j < K; j++) {
        printf("%f ", distances_sorted[j]);
    }
    printf("\n\n");


    /*============== CPU functions ==============*/

    // this knn_cpu function is not working properly
    // and i'm thinking to use different sorting algorithm on CPU to compare with GPU
    //  merge sort is necessary, we can add more sorting algorithm to compare
    auto started=std::chrono::high_resolution_clock::now();
    auto indices_sorted_CPU = new std::vector<std::pair<std::vector<float>, float>>();
    knn_cpu(indices_sorted_CPU,data_input_vec,data_query_vec);

    auto stopped =std::chrono::high_resolution_clock::now();

    /*============== Compare result ==============*/

//    QueryPerformanceCounter(&end_time);
//
    printf("============== CPU ==============\n\n");

    std::chrono::duration<double, std::milli> elapsed = stopped - started;
    std::cout << "CPU execution time\t\t\t:" << elapsed.count() <<"\n\n";

    // Print sorted indices and distances
//    printf("Find the indices of K nearest neighbors(CPU) : \n");
//    for (int j = 0; j < K; j++) {
//        printf("%f ", indices_sorted_CPU->at(j).second);
//    }
    printf("\n\n");

    printf("Find the distances of K nearest neighbors(CPU) : \n");
    for (int j = 0; j < K; j++) {
        printf("%f ", indices_sorted_CPU->at(j).second);
    }
    printf("\n\n");
//

    /*============== Free memory ==============*/

    // Free memory
    free(data_input);
    free(data_query);
    free(indices_sorted);
    free(distances_sorted);
    free(distances_sorted_CPU);
//    free(indices_sorted_CPU);
    free(indices);
    //free(distances_unsorted);
    hipFree(d_dis_temp);
    hipFree(d_ind_temp);
    hipFree(d_query);
    hipFree(d_input);
    hipFree(d_distances);
    hipFree(d_indices);

    return 0;
}
