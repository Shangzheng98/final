#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <cmath>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#define N 262144 // number of data points
#define D 40 // dimension of each data point
#define K 10 // number of nearest neighbors
#define BLOCK_SIZE_DIS 8 // number of threads per block for distance calculation
#define BLOCK_SIZE_KNN 1 // number of threads per block for knn calculation


//--------------------------------------------------------------------------------------//
//                                     CUDA Kernels                                     //
//--------------------------------------------------------------------------------------//

/**
 * This CUDA kernel calculates the Euclidean distance between a query point and a set of data points.
 *
 * input：dimension, num_points, data_input, query
 * @param dimension The dimension of each point (number of features).
 * @param num_points The number of data points to calculate distances for.
 * @param data_input An array containing the data points.
 * @param query An array containing the query point.
 *
 * output：distances
 * @param distances An array to store the calculated distances.
 *
 */
__global__ void euclidean_distance_kernel(float* distances, float* data_input, float* query, int dimension, int num_points) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points) {
        float sum = 0;
        for (int i = 0; i < dimension; i++) {
            float diff = data_input[idx * dimension + i] - query[i];
            sum += diff * diff;
        }
        distances[idx] = sqrt(sum);
    }
}

/**
 * This CUDA kernel use a merge sort to sort the distances and indices.
 * It is used in the main function to find the k nearest neighbors.
 *
 * input：num_points, ind_temp, dis_temp
 * @param num_points The number of distances to sort.
 * @param ind_temp An array temprarly storing the indices to pass to the next iteration.
 * @param dis_temp An array temprarly storing the distances to pass to the next iteration.
 *
 * output：indices, distances
 * @param indices An array to store the indices of the sorted distances.
 * @param distances An array to store the sorted distances.
 *
 * @NOTE: The number of threads per block must be 1. And the total number of the data points must be a power of 2.
 */
__global__ void mergeBlocks_kernel(int* indices, float* distances, int* ind_temp, float* dis_temp, int sortedsize) {
    int idx = blockIdx.x;

    //index for distances
    int index1 = idx * 2 * sortedsize;
    int endIndex1 = index1 + sortedsize;
    int index2 = endIndex1;
    int endIndex2 = index2 + sortedsize;
    int targetIndex = idx * 2 * sortedsize;

    //pesudo index for indices
    //let the indices divide and merge like distances
    int index1_ind = index1;
    int endIndex1_ind = endIndex1;
    int index2_ind = index2;
    int endIndex2_ind = endIndex2;
    int targetIndex_ind = targetIndex;

    int done = 0;
    while (!done)
    {
        //if the first block is not finished and the second block is not finished
        if ((index1 == endIndex1) && (index2 < endIndex2)) {
            dis_temp[targetIndex++] = distances[index2++];
            ind_temp[targetIndex_ind++] = indices[index2_ind++];
        }

        //if the second block is finished and the first block is not finished
        else if ((index2 == endIndex2) && (index1 < endIndex1)) {
            dis_temp[targetIndex++] = distances[index1++];
            ind_temp[targetIndex_ind++] = indices[index1_ind++];
        }

        //if the first block is smaller than the second block
        else if (distances[index1] < distances[index2]) {
            dis_temp[targetIndex++] = distances[index1++];
            ind_temp[targetIndex_ind++] = indices[index1_ind++];
        }

        //else
        else {
            dis_temp[targetIndex++] = distances[index2++];
            ind_temp[targetIndex_ind++] = indices[index2_ind++];
        }

        //if both blocks are finished, merge done.
        if ((index1 == endIndex1) && (index2 == endIndex2))
            done = 1;
    }
}

/**
* This CUDA kernel implement a swap function.
*
* input：a, b
* @param a The first number to swap.
* @param b The second number to swap.
*
* output：a, b
* @param a The first number after swapping.
* @param b The second number after swapping.
*
*/
__device__ void swap(float* a, float* b) {
    float temp = *a;
    *a = *b;
    *b = temp;
}

/**
* This CUDA kernel implement a bitonic sort.
*
* input：in, length, step, num_points
* @param in An array containing the data points.
* @param length The length that the array is divided into.
* @param step The step that the array is divided into.
* @param num_points The number of data points to sort.
*
* output：in
* @param in An array containing the sorted data points.
*
* @NOTE: The total number of the data points must be a power of 2.
*
*/
__global__ void Bitonic_Sort_kernel(float* in, int length, int step, int num_points) {
    unsigned int idx;
    idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < num_points) {
        int index = idx | step; //Find the pair index of the element we want to compare
        if ((idx & length) == 0) { //ascending
            if (in[idx] > in[index]) {
                swap(&in[idx], &in[index]);
            }
        }

        else { //descending
            if (in[idx] < in[index]) {
                swap(&in[idx], &in[index]);
            }
        }
    }
}



//--------------------------------------------------------------------------------------//
//                                     CPU functions                                    //
//--------------------------------------------------------------------------------------//


/**
 * This CPU function calculates the Euclidean distance between a query point and a set of data points.
 *
 * input：dimension, num_points, data_input, query
 * @param dimension The dimension of each point (number of features).
 * @param num_points The number of data points to calculate distances for.
 * @param data_input An array containing the data points.
 * @param query An array containing the query point.
 *
 * output：distances
 * @param distances An array to store the calculated distances.
 *
 */
float euclidean_distance_cpu(std::vector<float>& vec1, std::vector<float>& vec2) {
    float distance = 0.0;
    for (int j = 0; j < vec1.size(); j++) {
        float diff = vec1[j] - vec2[j];
        distance += diff * diff;
    }

    return sqrt(distance);
}

/**
 * This CPU function implements sorting the distances and indices.
 *
 */
void knn_cpu(std::vector<std::pair<std::vector<float>, float>>* distances, std::vector<std::vector<float>>& trainSet, std::vector<float>& queryData) {
    for (auto j : trainSet) {
        distances->push_back(std::make_pair(j, euclidean_distance_cpu(queryData, j)));
    }

    std::sort(distances->begin(), distances->end(), [](const std::pair<std::vector<float>, float>& p1, const std::pair<std::vector<float>, float>& p2) {
        return p1.second < p2.second;
        });
}

/**
 *This is a C++ class called "Timer" that allows timing of program execution.
 *It uses the <chrono> library to measure time with high resolution clock.
 *
 */
class Timer {
public:
    Timer() {
        stopped = started = std::chrono::high_resolution_clock::now();
    };

    Timer& start() {
        started = std::chrono::high_resolution_clock::now();
        return *this;
    }

    Timer& stop() {
        stopped = std::chrono::high_resolution_clock::now();
        return *this;
    }

    double elapsed() {
        if (started != stopped) {
            std::chrono::duration<double> elapsed = stopped - started;
            return elapsed.count();
        }
        return 0.0;
    }

    std::chrono::time_point<std::chrono::high_resolution_clock> started;
    std::chrono::time_point<std::chrono::high_resolution_clock> stopped;
};

//--------------------------------------------------------------------------------------//
//                                     Main function                                    //
//--------------------------------------------------------------------------------------//


/**
 * main function to see if GPU and CPU functions work and compare running time
 */
int main() {

    /*============== Initialize and Memory Allocated ==============*/

    // (HOST) Define and allocate memory for data input
    float* data_input;
    data_input = (float*)malloc(N * D * sizeof(float));

    // (HOST) Initialize data input
    for (int i = 0; i < N * D; i++) {
        data_input[i] = (float)rand() / (float)RAND_MAX;
    }

    // (HOST) Convert data_input to vector for CPU computation
    std::vector<std::vector<float> > data_input_vec(N, std::vector<float>(D));
    for (int i = 0; i < N * D; i++) {
        data_input_vec[i / D][i % D] = data_input[i];
    }

    /*
    //using this to test if distance is calculated correctly and if the sorting works
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < D; j++) {
            data_input[i * D + j] = i+1;
        }
    }

    std::vector<float> data_input_vec(data_input, data_input + N * D);
    */

    // (HOST) Define and allocate memory for query data
    float* data_query;
    data_query = (float*)malloc(D * sizeof(float));

    // (HOST) Initialize query data
    for (int i = 0; i < D; i++) {
        data_query[i] = (float)rand() / (float)RAND_MAX;
    }

    // (HOST) Convert data_query to vector for CPU computation
    std::vector<float> data_query_vec(data_query, data_query + D);

    /*
    //using this to test if distance is calculated correctly and if the sorting works
    for (int i = 0; i <D; i++) {
        data_query[i] = -1;
    }
    */

    // (DEVICE) Copy input data to device memory
    float* d_input;
    hipMalloc(&d_input, N * D * sizeof(float));
    hipMemcpy(d_input, data_input, N * D * sizeof(float), hipMemcpyHostToDevice);


    // (DEVICE) Copy query data to device memory
    float* d_query;
    hipMalloc(&d_query, D * sizeof(float));
    hipMemcpy(d_query, data_query, D * sizeof(float), hipMemcpyHostToDevice);

    // (DEVICE) Allocate memory for temporary distances and indices
    float* d_dis_temp;
    hipMalloc(&d_dis_temp, N * sizeof(float));
    int* d_ind_temp;
    hipMalloc(&d_ind_temp, N * sizeof(int));

    // (DEVICE) Allocate memory for indices
    int* d_indices;
    hipMalloc(&d_indices, N * sizeof(int));

    /*
    // Allocate memory for unsorted distances
    float* distances_unsorted = (float*)malloc(N * sizeof(float));
    hipMemcpy(distances_unsorted, d_distances, N * sizeof(float), hipMemcpyDeviceToHost);
    */

    // (DEVICE) Initialize indices and copy to device memory
    int* indices = (int*)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++) {
        indices[i] = i;
    }
    hipMemcpy(d_indices, indices, N * sizeof(int), hipMemcpyHostToDevice);

    // (DEVICE) Allocate memory for distances
    float* d_distances_merge;
    float* d_distances_bitonic;
    hipMalloc(&d_distances_merge, N * sizeof(float));
    hipMalloc(&d_distances_bitonic, N * sizeof(float));

    // (HOST) Allocate memory for sorted distances and indices computed by GPU
    float* distances_sorted_merge = (float*)malloc(N * sizeof(float));
    float* distances_sorted_bitonic = (float*)malloc(N * sizeof(float));
    int* indices_sorted = (int*)malloc(N * sizeof(int));

    // (HOST) Allocate memory for sorted distances and indices computed by CPU
    float* distances_sorted_CPU = (float*)malloc(N * sizeof(float));
    auto indices_sorted_CPU_vec = new std::vector<float>();


    /*============== GPU functions ==============*/

    /*-----Merge Sort-----*/
    hipEvent_t start1, stop1;
    checkCudaErrors(hipEventCreate(&start1));
    checkCudaErrors(hipEventCreate(&stop1));

    checkCudaErrors(hipEventRecord(start1));

    // Compute distances between current query point and data input
    euclidean_distance_kernel << <(N + BLOCK_SIZE_DIS - 1) / BLOCK_SIZE_DIS, BLOCK_SIZE_DIS >> > 
    (d_distances_merge, d_input, d_query, D, N);

    // Sort distances and indices
    int blocks_merge = (N + BLOCK_SIZE_KNN - 1) / BLOCK_SIZE_KNN / 2;
    int sortedsize = BLOCK_SIZE_KNN;
    while (blocks_merge > 0) {
        mergeBlocks_kernel << <blocks_merge, 1 >> > (d_indices, d_distances_merge, d_ind_temp, d_dis_temp, sortedsize);
        hipMemcpy(d_distances_merge, d_dis_temp, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_indices, d_ind_temp, N * sizeof(int), hipMemcpyDeviceToDevice);
        blocks_merge /= 2;
        sortedsize *= 2;
    }

    checkCudaErrors(hipEventRecord(stop1));

    // Copy sorted distances and indices to host memory
    hipMemcpy(indices_sorted, d_indices, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(distances_sorted_merge, d_distances_merge, N * sizeof(float), hipMemcpyDeviceToHost);

    //    std::vector<float> distances_sorted_vec_GPU(distances_sorted, distances_sorted + N);

    checkCudaErrors(hipEventSynchronize(stop1));

    /*-----Bitonic Sort-----*/
    hipEvent_t start2, stop2;
    checkCudaErrors(hipEventCreate(&start2));
    checkCudaErrors(hipEventCreate(&stop2));

    checkCudaErrors(hipEventRecord(start2));


    // Compute distances between current query point and data input
    euclidean_distance_kernel << <(N + BLOCK_SIZE_DIS - 1) / BLOCK_SIZE_DIS, BLOCK_SIZE_DIS >> > 
    (d_distances_bitonic, d_input, d_query, D, N);

    int threads = 1024;
    int blocks_bitonic = (N + threads - 1) / threads;
    for (int k = 2; k <= N; k <<= 1) {
        for (int j = k >> 1; j > 0; j = j >> 1) {
            Bitonic_Sort_kernel << <blocks_bitonic, threads >> > (d_distances_bitonic, k, j, N);
        }
    }

    checkCudaErrors(hipEventRecord(stop2));

    // Copy sorted distances and indices to host memory
    //hipMemcpy(indices_sorted, d_indices, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(distances_sorted_bitonic, d_distances_bitonic, N * sizeof(float), hipMemcpyDeviceToHost);

    //    std::vector<float> distances_sorted_vec_GPU(distances_sorted, distances_sorted + N);

    checkCudaErrors(hipEventSynchronize(stop2));


    printf("============== GPU ==============\n\n");

    // Print query point
    printf("Query point : ");
    for (int i = 0; i < D; i++) {
        printf("%f ", data_query[i]);
    }
    printf("\n\n");

    // Compute and print kernel execution time for merge sort
    float kernel_time1;
    checkCudaErrors(hipEventElapsedTime(&kernel_time1, start1, stop1));
    printf("Kernel execution time of merge sort\t\t\t: %f ms\n\n", kernel_time1);



    // Print sorted indices and distances
    printf("Find the indices of K nearest neighbors(GPU, merge) : \n");
    for (int j = 0; j < K; j++) {
        printf("%d ", indices_sorted[j]);
    }
    printf("\n\n");

    printf("Find the distances of K nearest neighbors(GPU, merge) : \n");
    for (int j = 0; j < K; j++) {
        printf("%f ", distances_sorted_merge[j]);
    }
    printf("\n\n");

    // Compute and print kernel execution time for bitonic sort
    float kernel_time2;
    checkCudaErrors(hipEventElapsedTime(&kernel_time2, start2, stop2));
    printf("Kernel execution time of bitonic sort\t\t\t: %f ms\n\n", kernel_time2);

    printf("Find the distances of K nearest neighbors(GPU, bitonic) : \n");
    for (int j = 0; j < K; j++) {
        printf("%f ", distances_sorted_bitonic[j]);
    }
    printf("\n\n");


    /*============== CPU functions ==============*/

    auto started = std::chrono::high_resolution_clock::now();
    auto indices_sorted_CPU = new std::vector<std::pair<std::vector<float>, float>>();
    knn_cpu(indices_sorted_CPU, data_input_vec, data_query_vec);

    auto stopped = std::chrono::high_resolution_clock::now();

    /*============== Compare result ==============*/

    printf("============== CPU ==============\n\n");

    std::chrono::duration<double, std::milli> elapsed = stopped - started;
    std::cout << "CPU execution time\t\t\t:" << elapsed.count() << "\n\n";

    // Print sorted indices and distances
//    printf("Find the indices of K nearest neighbors(CPU) : \n");
//    for (int j = 0; j < K; j++) {
//        printf("%f ", indices_sorted_CPU->at(j).second);
//    }
    printf("\n\n");

    printf("Find the distances of K nearest neighbors(CPU) : \n");
    for (int j = 0; j < K; j++) {
        printf("%f ", indices_sorted_CPU->at(j).second);
    }
    printf("\n\n");
    //

    /*============== Free memory ==============*/

    checkCudaErrors(hipEventDestroy(start1));
    checkCudaErrors(hipEventDestroy(stop1));
    checkCudaErrors(hipEventDestroy(start2));
    checkCudaErrors(hipEventDestroy(stop2));

    // Free memory
    free(data_input);
    free(data_query);
    free(indices_sorted);
    //free(distances_sorted);
    free(distances_sorted_merge);
    free(distances_sorted_bitonic);
    free(distances_sorted_CPU);
    //free(indices_sorted_CPU);
    free(indices);
    //free(distances_unsorted);
    hipFree(d_dis_temp);
    hipFree(d_ind_temp);
    hipFree(d_query);
    hipFree(d_input);
    //hipFree(d_distances);
    hipFree(d_distances_merge);
    hipFree(d_distances_bitonic);
    hipFree(d_indices);

    return 0;
}
